
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <algorithm>
#include <iterator>
#include <vector>
#include <functional>

__global__ void add(int* in, int* out) {
	int idx = threadIdx.x;
	out[idx] = in[idx] * in[idx];
}


int main(int argc, char* argv[]) {

	std::vector<int> dd(512);
	std::iota(std::begin(dd), std::end(dd), 1);

	auto sz = 512 * sizeof(int);
	int* din;
	int* dout;

	hipMalloc( (void**)&din, sz );
	hipMalloc( (void**)&dout, sz );

	hipMemcpy(din, &(dd.data()[0]), sz, hipMemcpyHostToDevice);
	add<<<1, 512>>>(din, dout);
	
	std::vector<int> oo(512, 0);
	hipMemcpy(&(oo.data()[0]), dout, sz, hipMemcpyDeviceToHost);

	hipFree( din );
	hipFree( dout );

	std::copy(std::begin(oo), std::end(oo), std::ostream_iterator<int>(std::cout, " "));
	std::cout << "\n";

	return 0;
}
